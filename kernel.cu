//N-Body Physics Simulation
//Randomly generates and displays an N-Body system
//Code from Nvidia's GPU Gems 3 Chapter 31

#ifndef __CUDACC__
#define __CUDACC__
#endif
#include "hip/hip_runtime_api.h"
#include "hip/hip_runtime.h"

#include "hip/device_functions.h"


#include <stdio.h>
#include <cmath>
#include <cstdlib>
#include <ctime>
#include <iostream>
using namespace std;

//Gravitational constant
#define G 1

//Time interval
#define dx 0.001

//N, number of bodies
const unsigned int N = 100;

//p, number of blocks
const unsigned int p = 5;

//Upper bounds for location
const float upperX = 100.0;
const float upperY = 100.0;
const float upperZ = 100.0;

//Lower bounds for location
const float lowerX = -100.0;
const float lowerY = -100.0;
const float lowerZ = -100.0;

//Mass bounds
const float upperMass = 100.0;
const float lowerMass = 1.0;

//Interaction between two bodies
//Integrate using leapfrog-Verlet integrator
//a ~= G * summation(1, N, m[j]*r[i][j] / (r^2 + E^2)^3/2
//G = 1 for simplicity
__device__ float3 bobyBodyInteraction(float4 b1, float4 b2, float3 a1) {
	//Distance between bodies
	float3 d;
	//3 FLOPS
	d.x = b2.x - b1.x;
	d.y = b2.y - b1.y;
	d.z = b2.z - b1.z;

	//Square distance
	//6 FLOPS
	float square = d.x * d.x + d.y * d.y + d.z * d.z;

	//Cube
	//2 FLOPS
	float cube = square * square * square;

	//Invert and sqrt
	//2 FLOPS
	float invert = 1.0f / sqrtf(cube);

	//Calculate s
	float s = b2.w * invert;


	//Calculate a
	a1.x += d.x * s * G;
	a1.y += d.y * s * G;
	a1.z += d.z * s * G;
	return a1;

}

//Calculate all interactions within a tile
__device__ float3 tile_calculation(float4 pos, float3 a) {
	int i;
	//Used shared memory
	extern __shared__ float4 sPos[];
	for (i = 0; i < blockDim.x; i++) {
		a = bobyBodyInteraction(pos, sPos[i], a);
	}
	return a;
}

//Calculate acceleration for p bodies with p threads resulting from N interactions
__global__ void calculate_forces(void *devX, void *devA) {
	//Declare shared position
	extern __shared__ float4 sPos[];

	//Get from memory
	float4 *globalX = (float4 *)devX;
	float4 *globalA = (float4 *)devA;

	//Initialize position
	float4 pos;

	//Initialize variables
	int i, tile;
	float3 acc = { 0.0f, 0.0f, 0.0f };

	//Get index
	int getId = blockIdx.x * blockDim.x + threadIdx.x;

	//Get position
	pos = globalX[getId];

	//Calculate N bodies
	for (i = 0, tile = 0; i < N; i += p, tile++) {
		//Get index
		int idx = tile * blockDim.x + threadIdx.x;

		//Update position
		sPos[threadIdx.x] = globalX[idx];

		//Barrier
		__syncthreads();

		//Get acceleration
		acc = tile_calculation(pos, acc);

		//Barrier
		__syncthreads();
	}

	//Save in global memory
	float4 acc4 = { acc.x, acc.y, acc.z, 0.0f };
	globalA[getId] = acc4;

}

int main()
{
	//Generate N random bodies with locations defined by bounds
	float4 *h_s = (float4*)malloc(N * sizeof(float4));
	float3 *h_v = (float3*)malloc(N * sizeof(float3));
	float3 *h_a = (float3*)malloc(N * sizeof(float3));

	srand(time(NULL));
	for (int i = 0; i < N; i++) {
		h_s[i].x = ((float)rand() / RAND_MAX) * (upperX - lowerX) + lowerX;
		h_s[i].y = ((float)rand() / RAND_MAX) * (upperY - lowerY) + lowerY;
		h_s[i].z = ((float)rand() / RAND_MAX) * (upperZ - lowerZ) + lowerZ;
		h_s[i].w = ((float)rand() / RAND_MAX) * (upperMass - lowerMass) + lowerMass;

		//No initial velocity or acceleration 
		h_v[i].x = 0;
		h_v[i].y = 0;
		h_v[i].z = 0;
		h_a[i].x = 0;
		h_a[i].y = 0;
		h_a[i].z = 0;
	}

	//Create memory on device
	float4 *d_s;
	hipMalloc(&d_s, N * sizeof(float4));
	float3 *d_v;
	hipMalloc(&d_v, N * sizeof(float3));
	float3 *d_a;
	hipMalloc(&d_a, N * sizeof(float3));

	//Copy memory to device
	hipMemcpy(d_s, h_s, N * sizeof(float4), hipMemcpyHostToDevice);
	hipMemcpy(d_v, h_v, N * sizeof(float3), hipMemcpyHostToDevice);
	hipMemcpy(d_a, h_a, N * sizeof(float3), hipMemcpyHostToDevice);

	//Calculate accelerations
	calculate_forces <<<1, N>>>(d_s, d_a);

	//Return accelerations
	hipMemcpy(h_a, d_a, N * sizeof(float3), hipMemcpyDeviceToHost);

	//Print accelerations
	for (unsigned int i = 0; i < N; i++) {
		cout << h_a[i].x << ", " << h_a[i].y << ", " << h_a[i].z << endl;
	}

    return 0;
}